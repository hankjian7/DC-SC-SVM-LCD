#include "hip/hip_runtime.h"
#include "BruteForceKNN_raft.hpp"
#include "GlobalDefine.hpp"
#include <chrono>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/host_mdarray.hpp>
#include <raft/matrix/copy.cuh>
#include <raft/core/resource/hip_stream.hpp>
#include <raft/neighbors/brute_force.cuh>
#include <memory>
__global__ void updateRowsKernel(
    float* codebook,        // 完整的 codebook (65536x128)
    const float* new_data,  // 新的向量數據
    const uint32_t* indices,// 要更新的索引
    int num_rows,          // codebook 的總行數 (65536)
    int num_updates,       // 要更新的行數
    int dims              // 向量維度 (128)
) {
    // 計算當前線程要處理的更新索引和維度
    int update_idx = blockIdx.x;  // 處理哪一個更新
    int dim_idx = threadIdx.x;    // 處理向量的哪一個維度
    
    // 確保線程索引在有效範圍內
    if (update_idx < num_updates && dim_idx < dims) {
        // 獲取要更新的 codebook 行索引
        uint32_t codebook_row = indices[update_idx];
        
        // 確保索引有效
        if (codebook_row < num_rows) {
            // 更新數據
            // codebook[codebook_row * dims + dim_idx] = new_data[update_idx * dims + dim_idx];
            atomicExch(
                &codebook[codebook_row * dims + dim_idx],
                new_data[update_idx * dims + dim_idx]
            );
        }
    }
}

void updateCodebookRows(
    raft::device_resources& dev_resources,
    float* new_vectors,          
    const uint32_t* update_indices,  
    int num_updates,            
    int dims,                   
    MemoryPreallocation& memory // 包含 codebook_device (65536x128)
) {
    // 1. 創建並複製新向量到 device
    auto new_vectors_device = raft::make_device_matrix<float, uint32_t>(
        dev_resources, num_updates, dims
    );
    
    // 創建臨時 host matrix 並複製數據
    auto new_vectors_host = raft::make_host_matrix<float, uint32_t>(
        dev_resources, num_updates, dims
    );
    std::memcpy(
        new_vectors_host.data_handle(),
        new_vectors,
        num_updates * dims * sizeof(float)
    );
    
    // 複製到 device
    raft::copy(dev_resources, new_vectors_device.view(), new_vectors_host.view());

    // 2. 創建並複製索引到 device
    auto indices_device = raft::make_device_vector<uint32_t, uint32_t>(
        dev_resources, num_updates
    );
    
    // 創建臨時 host vector 並複製索引
    auto indices_host = raft::make_host_vector<uint32_t, uint32_t>(
        dev_resources, num_updates
    );
    for (int i = 0; i < num_updates; i++) {
        indices_host.data_handle()[i] = static_cast<uint32_t>(update_indices[i]);
    }
    
    // 複製到 device
    raft::copy(dev_resources, indices_device.view(), indices_host.view());

    // 3. 配置並啟動 CUDA kernel
    int total_rows = memory.codebook_device.extent(0);  // 65536
    
    // 配置線程塊和網格
    dim3 block(128);  // 每個線程處理一個維度
    dim3 grid(num_updates);  // 每個塊處理一個更新
    
    // 獲取 CUDA stream
    auto stream = raft::resource::get_cuda_stream(dev_resources);
    
    // 啟動 kernel
    updateRowsKernel<<<grid, block, 0, stream>>>(
        memory.codebook_device.data_handle(),
        new_vectors_device.data_handle(),
        indices_device.data_handle(),
        total_rows,
        num_updates,
        dims
    );
    
    // 同步確保更新完成
    hipStreamSynchronize(stream);
}
void codebookToDevice(
	raft::device_resources& dev_resources, 
	float* codebook, 
	int numVectors, 
	int dims, 
	MemoryPreallocation& memory) 
{
	raft::host_matrix<float, uint32_t> codebook_host = raft::make_host_matrix<float, uint32_t>(dev_resources, numVectors, dims);
	std::memcpy(codebook_host.data_handle(), codebook, numVectors * dims * sizeof(float));
	raft::copy(dev_resources, memory.codebook_device.view(), codebook_host.view());
}
void queriesToDevice(
	raft::device_resources& dev_resources, 
	float* queries, 
	int numQueries, 
	int dims, 
	MemoryPreallocation& memory) 
{
	std::memcpy(memory.queries_host.data_handle(), queries, numQueries * dims * sizeof(float));
	raft::copy(dev_resources, memory.queries_device.view(), memory.queries_host.view());
}
void bruteForceKNN(raft::device_resources& dev_resources, Args& args, MemoryPreallocation& memory)
{
	// auto codebook_host = raft::make_host_matrix<float>(dev_resources, args.numVectors, args.dims);
	// auto queries_host = raft::make_host_matrix<float>(dev_resources, args.numQueries, args.dims);
	// std::memcpy(memory.codebook_host.data_handle(), args.codebook, args.numVectors * args.dims * sizeof(float));
    // std::memcpy(memory.queries_host.data_handle(), args.queries, args.numQueries * args.dims * sizeof(float));
	// auto copy1_start = std::chrono::high_resolution_clock::now();
    // raft::copy(dev_resources, memory.codebook_device.view(), memory.codebook_host.view());
    // raft::copy(dev_resources, memory.queries_device.view(), memory.queries_host.view());
	// auto copy1_end = std::chrono::high_resolution_clock::now();
	// std::cout << "copy1 time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(copy1_end - copy1_start).count() << " ms" << std::endl;
	raft::device_matrix_view<const float, uint32_t> codebook_view = memory.codebook_device.view();
	raft::device_matrix_view<const float, uint32_t> queries_view = memory.queries_device.view();
	auto neighbors_device = raft::make_device_matrix<uint32_t, uint32_t>(dev_resources, args.numQueries, args.k);
	auto distances_device = raft::make_device_matrix<float, uint32_t>(dev_resources, args.numQueries, args.k);
	raft::device_matrix_view<uint32_t, uint32_t> neighbors_view = neighbors_device.view();
	raft::device_matrix_view<float, uint32_t> distances_view = distances_device.view();
    // Perform brute force KNN search.
	raft::resource::sync_stream(dev_resources);
    // auto metric = raft::distance::DistanceType::L2Unexpanded; // Using L2 distance
	//auto knn_start = std::chrono::high_resolution_clock::now();
    raft::neighbors::brute_force::knn(
        dev_resources,
        std::vector{codebook_view},
        queries_view,
        neighbors_view,
        distances_view,
        raft::distance::DistanceType::L2Unexpanded
    );
	//auto knn_end = std::chrono::high_resolution_clock::now();
	//std::cout << "knn time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(knn_end - knn_start).count() << " ms" << std::endl;
	//print_results(dev_resources, neighbors_view, distances_view);
    // The call to brute_force::knn is asynchronous. Before accessing the data, sync by calling
	auto neighbors_host = raft::make_host_matrix<uint32_t, uint32_t>(args.numQueries, args.k);
	auto distances_host = raft::make_host_matrix<float, uint32_t>(args.numQueries, args.k);
	hipStream_t stream = raft::resource::get_cuda_stream(dev_resources);
	raft::copy(neighbors_host.data_handle(), neighbors_device.view().data_handle(), neighbors_device.view().size(), stream);
	raft::copy(distances_host.data_handle(), distances_device.view().data_handle(), distances_device.view().size(), stream);
	raft::resource::sync_stream(dev_resources, stream);
	std::memcpy(args.outIndices, neighbors_host.data_handle(), neighbors_device.size() * sizeof(uint32_t));
    std::memcpy(args.outDistances, distances_host.data_handle(), distances_device.size() * sizeof(float));
}
// #include <raft/core/device_resources.hpp>
// #include <raft/core/resource/hip_stream.hpp>
// #include <raft/core/device_mdarray.hpp>
// #include <raft/core/host_mdarray.hpp>
// // #include <raft/core/device_mdspan.hpp>
// #include <raft/neighbors/brute_force.cuh>
// // #include <raft/util/cudart_utils.hpp>

// #include <cstdint>
// #include <optional>
// #include <iostream>
// #include <Eigen/Dense>
// #include <vector>
// #include <cmath>
// #include <chrono>
// #include <random>
// typedef Eigen::Matrix<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> MatrixXfR;
// typedef Eigen::Matrix<int, Eigen::Dynamic, Eigen::Dynamic, Eigen::RowMajor> MatrixXiR;

// Custom structure to hold arguments


// void knn_raft(Args& args) {
// 	using namespace raft::neighbors;
// 	// Initialize RAFT device resources
// 	raft::device_resources handle;

// 	// Create device buffers for input and output data
// 	auto start = std::chrono::high_resolution_clock::now();
// 	auto index = raft::make_readonly_temporary_device_buffer<
// 	const float,
// 	int,
// 	raft::col_major>(
// 	handle,
// 	const_cast<float*>(
// 			reinterpret_cast<const float*>(args.codebook)),
// 	raft::matrix_extent<int>(args.numVectors, args.dims));
//     // auto index = raft::make_device_matrix<float, uint32_t, raft::col_major>(
//     //     handle,
//     //     raft::matrix_extent<int>(args.numVectors, args.dims));
//     // raft::copy(index.data_handle(), args.codebook, args.numVectors * args.dims, handle.get_stream());
// 	auto end = std::chrono::high_resolution_clock::now();
// 	std::cout << "index time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;

// 	start = std::chrono::high_resolution_clock::now();
// 	auto search = raft::make_readonly_temporary_device_buffer<
// 			const float,
// 			int,
// 			raft::col_major>(
// 			handle,
// 			const_cast<float*>(
// 					reinterpret_cast<const float*>(args.queries)),
// 			raft::matrix_extent<int>(args.numQueries, args.dims));
// 	end = std::chrono::high_resolution_clock::now();
// 	std::cout << "search time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;
// 	start = std::chrono::high_resolution_clock::now();
// 	auto inds =
// 			raft::make_writeback_temporary_device_buffer<int, uint32_t>(
// 					handle,
// 					reinterpret_cast<int*>(args.outIndices),
// 					raft::matrix_extent<int>(args.numQueries, args.k));
// 	end = std::chrono::high_resolution_clock::now();
// 	std::cout << "inds time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;
// 	start = std::chrono::high_resolution_clock::now();	
// 	auto dists =
// 			raft::make_writeback_temporary_device_buffer<float, uint32_t>(
// 					handle,
// 					reinterpret_cast<float*>(args.outDistances),
// 					raft::matrix_extent<int>(args.numQueries, args.k));
// 	end = std::chrono::high_resolution_clock::now();
// 	std::cout << "dists time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;					
	
// 	start = std::chrono::high_resolution_clock::now();
// 	std::vector<raft::device_matrix_view<
// 			const float,
// 			int,
// 			raft::col_major>>
// 			index_vec = {index.view()};
// 	end = std::chrono::high_resolution_clock::now();
// 	std::cout << "index_vec time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;
// 	// Perform kNN search using RAFT
// 	start = std::chrono::high_resolution_clock::now();
// 	brute_force::knn(
// 			handle,
// 			index_vec,
// 			search.view(),
// 			inds.view(),
// 			dists.view(),
// 			raft::distance::DistanceType::L2Unexpanded,  // Using L2SqrtUnexpanded as distance metric
// 			args.metricArg);
// 	end = std::chrono::high_resolution_clock::now();
// 	std::cout << "Inner time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << " ms" << std::endl;
// 	// Synchronize device resources
// 	handle.sync_stream();
// }

// inline void preallocate_memory(raft::device_resources& dev_resources, Args& args, MemoryPreallocation& memory) {
// 	memory.codebook_device = raft::make_device_matrix<float, uint32_t>(dev_resources, args.numVectors, args.dims);
// 	memory.queries_device = raft::make_device_matrix<float, uint32_t>(dev_resources, args.numQueries, args.dims);
// 	memory.neighbors_device = raft::make_device_matrix<int, uint32_t>(dev_resources, args.numQueries, args.k);
// 	memory.distances_device = raft::make_device_matrix<float, uint32_t>(dev_resources, args.numQueries, args.k);
// 	memory.codebook_host = raft::make_host_matrix<float>(dev_resources, args.numVectors, args.dims);
// 	memory.queries_host = raft::make_host_matrix<float>(dev_resources, args.numQueries, args.dims);
// 	memory.neighbors_host = raft::make_host_matrix<int>(dev_resources, args.numQueries, args.k);
// 	memory.distances_host = raft::make_host_matrix<float>(dev_resources, args.numQueries, args.k);
// }

// struct Args {
//     int dims;
//     int numVectors;
//     int numQueries;
//     int k;
//     float* codebook;
//     float* queries;
//     int* outIndices;
//     float* outDistances;
// };
// struct MemoryPreallocation {
// 	raft::device_matrix<float, uint32_t> codebook_device;
// 	raft::device_matrix<float, uint32_t> queries_device;
// 	raft::host_matrix<float, uint32_t> codebook_host;
//     raft::host_matrix<float, uint32_t> queries_host;

// 	MemoryPreallocation(raft::device_resources& dev_resources, Args const& args)
// 	: codebook_device(raft::make_device_matrix<float, uint32_t>(dev_resources, args.numVectors, args.dims)),
// 		queries_device(raft::make_device_matrix<float, uint32_t>(dev_resources, args.numQueries, args.dims)),
// 		codebook_host(raft::make_host_matrix<float, uint32_t>(dev_resources, args.numVectors, args.dims)),
//         queries_host(raft::make_host_matrix<float, uint32_t>(dev_resources, args.numQueries, args.dims)) {}
// };
// Function to initialize RAFT resources and perform kNN search


// int main() {
//     Args args;
//     args.dims = 128;
//     args.numVectors = 65536;
//     args.numQueries = 512;
//     args.k = 5;
//     std::mt19937 gen(0); // Standard mersenne_twister_engine seeded with 0
//     std::uniform_real_distribution<> dis(0.0, 1.0);

//     // Generate random data for demonstration
//     MatrixXfR codebook(args.numVectors, args.dims);
//     MatrixXfR queries(args.numQueries, args.dims);

//     for (int i = 0; i < args.numVectors; i++) {
//         for (int j = 0; j < args.dims; j++) {
//             //codebook(i, j) = i * args.dims + j;
//             codebook(i, j) = dis(gen);
//         }
//     }

//     for (int i = 0; i < args.numQueries; i++) {
//         for (int j = 0; j < args.dims; j++) {
//             //queries(i, j) = i * args.dims + j;
// 			queries(i, j) = dis(gen);
//         }
//     }
// 	//print_matrix(codebook);
// 	//print_matrix(queries);
//     args.codebook = codebook.data();
//     args.queries = queries.data();
//     args.outIndices = new int[args.numQueries * args.k];
//     args.outDistances = new float[args.numQueries * args.k];
// 	raft::device_resources dev_resources;
//     MemoryPreallocation memory(dev_resources, args);
// 	// memory.codebook_device = raft::make_device_matrix<float, uint32_t>(dev_resources, args.numVectors, args.dims);
// 	// memory.queries_device = raft::make_device_matrix<float, uint32_t>(dev_resources, args.numQueries, args.dims);
// 	// memory.neighbors_device = raft::make_device_matrix<int, uint32_t>(dev_resources, args.numQueries, args.k);
// 	// memory.distances_device = raft::make_device_matrix<float, uint32_t>(dev_resources, args.numQueries, args.k);

// 	// Perform kNN search using RAFT
//     auto outter_start = std::chrono::high_resolution_clock::now();
//     bruteForceKNN(dev_resources, args, memory);
//     auto outter_end = std::chrono::high_resolution_clock::now();
//     std::cout << "Outer time taken: " << std::chrono::duration_cast<std::chrono::milliseconds>(outter_end - outter_start).count() << " ms" << std::endl;

//     // Map the output pointers to Eigen matrices
//     Eigen::Map<MatrixXiR> inds_map(args.outIndices, args.numQueries, args.k);
//     Eigen::Map<MatrixXfR> dists_map(args.outDistances, args.numQueries, args.k);

//     // Output results
//     // std::cout << "Indices:\n" << inds_map << "\n";
//     // std::cout << "Distances:\n" << dists_map << "\n";

//     delete[] args.outIndices;
//     delete[] args.outDistances;

//     return 0;
// }